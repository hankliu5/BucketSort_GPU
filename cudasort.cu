#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <vector>
#include <ctime>
#include <string>
#include <vector>
#include <iterator>
#include <algorithm>
using namespace std;

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif

__global__ void gpu_sort(float *input, int *output, int* step) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  output[index] = __float2int_rd(input[index] / *step);
}

int cuda_sort(int number_of_elements, float *a, int step)
{
  const int NUM_BUCKETS = 6;
  float *d_in;
  int *d_out;
  int *out = (int *) malloc(sizeof(float) * number_of_elements);
  int *d_step;
  vector<float> buckets[NUM_BUCKETS];
  hipMalloc(&d_in, sizeof(float) * number_of_elements);
  hipMalloc(&d_out, sizeof(int) * number_of_elements);
  hipMalloc(&d_step, sizeof(int) * 1);

  hipMemcpy(d_in, a, sizeof(float) * number_of_elements, hipMemcpyHostToDevice);
  hipMemcpy(d_step, &step, sizeof(int) * 1, hipMemcpyHostToDevice);

  gpu_sort<<<number_of_elements/THREADS, THREADS>>>(d_in, d_out, d_step);
  hipMemcpy(out, d_out, sizeof(int) * number_of_elements, hipMemcpyDeviceToHost);
  for (int i = 0; i < number_of_elements; i++) {
    buckets[out[i]].push_back(a[i]);
  }
  for (int i = 0; i < NUM_BUCKETS; i++) {
    thrust::device_vector<float> d_vec = buckets[i];
    thrust::sort(d_vec.begin(), d_vec.end());
    thrust::copy(d_vec.begin(), d_vec.end(), buckets[i].begin());
  } 
  int index = 0;
  for (int i = 0; i < NUM_BUCKETS; i++)
  {
      for (vector<float>::iterator it = buckets[i].begin(); it != buckets[i].end(); it++)
      {
          a[index] = *it;
          index++;
      }
  }
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_step);
  free(out);
  return 0;
}

#ifdef __cplusplus
}
#endif
